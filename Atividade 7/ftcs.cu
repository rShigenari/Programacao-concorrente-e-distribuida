#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

/* constantes numericas para calculo do metodo ftcs */
#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T  0.01
#define kappa 0.000045


typedef long long int lli;

__global__ void funcaoCalcula(double *u, double *prev, double kap, double dt, double dx, int s){
	lli i =  blockDim.x * blockIdx.x + threadIdx.x;
	if(i>0 && i<(s-1))
		u[i] = prev[i] + kap*dt/(dx*dx)*(prev[i-1]-2*prev[i]+prev[i+1]);
}

__global__ void funcaoContorno(double *u, int n){
	lli i =  blockDim.x * blockIdx.x + threadIdx.x;
	if(i==1)
        u[0] = u[n] = 0.0;
}

int main(){
    /* variaveis do host */
	double t = 0.0, x = dx,  tmiliCuda;
	double  *u, *u_prev, *temp;
	lli i, n, maior, tam_blocos, tam;
    struct timeval inicioCuda, fimCuda;

	double *aux, *u_prev_prev; //variaveis do device


	n = tam/dx; //quantidade de pontos


   	 tam_blocos = ceil((n+1)/512);// calcula quantidade de blocos 

	/* aloca vetores no host */
    tam = (n+1)*sizeof(double);
	u = (double *) malloc(tam);
	u_prev = (double *) malloc(tam);

    /* preenche vetor u_prev no host */
	for(i=1; i<n; i++){
		if(x<=0.5) 
			u_prev[i] = 200.0*x;
		else        
			u_prev[i] = 200.0*(1.0-x);
		x += dx;
	}

    /* aloca vetores no device */
	hipMalloc((void **)&aux, tam);
	hipMalloc((void **)&u_prev_prev, tam);


	hipMemcpy(u_prev_prev, u_prev, tam, hipMemcpyHostToDevice); //copia valores do vetor u_prev do host para o u_prev_prev do device

    gettimeofday(&inicioCuda,NULL); //conta o tempo inicial do processamento para a medida de desempenho

	while(t<T){
		funcaoCalcula<<<tam_blocos, 512>>>(u_ua, u_prev_prev, kappa, dt, dx, n+1); //processamento paralelo no device	
      	        hipDeviceSynchronize(); //sincroniza o processamento no device para que possa realizar a troca de ponteiros antes da proxima iteracao
		funcaoContorno<<<tam_blocos,512>>>(aux, n);//condicao de contorno
		temp = aux; // realiza a troca de ponteiros, atualizando a proxima iteracao
		aux = u_prev_prev;
	    	u_prev_prev = temp;

	   	t += dt; //atualiza variavel de controle da estrutura de repeticao
	}
   	gettimeofday(&fimCuda,NULL);//obtem o tempo final do processamento
	hipMemcpy(u, aux, tam, hipMemcpyDeviceToHost);//copia os valores do vetor aux para o vetor u do host

	maior= 0; //acha o maior valor e a posicao do vetor
	for(i=1; i<n+1; ++i)
		if(u[i] > u[maior])
            maior= i;
    
    tmiliCuda = (double)(1000.0*(fimCuda.tv_sec-inicioCuda.tv_sec)+(fimCuda.tv_usec-inicioCuda.tv_usec)/1000.0);//calcula tempo de processamento

    printf("FTCS :: CUDA(CPU+GPU)\n");
    printf("Maximo valor de u[%lld] = %g\n", maior, u[maior]);
    printf("%lld Pontos e %lld Iteracoes\n", (n+1), (lli)(T/dt));
    printf("%lld Blocos e %lld Numero de threads por bloco\n", tam_blocos, (lli)512);
    printf("Tempo =  %.3lf ms\n", tmiliCuda);

    free(u); //libera memoria no host
    free(u_prev);
    u = NULL;
    u_prev = NULL;

    hipFree(aux);//libera memoria no device
    hipFree(u_prev_prev);

    return 0;
}
